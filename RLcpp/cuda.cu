#include "hip/hip_runtime.h"
#include <algorithm>
#include <memory>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>

// thrust
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/inner_product.h>
#include <thrust/adjacent_difference.h>

// RL
#include <RLcpp/thrustUtils.h>
#include <RLcpp/DeviceVector.h>

template <typename T>
class DeviceVector {
  public:
    virtual ~DeviceVector() {}
    virtual T* data() = 0;
    virtual T const* data() const = 0;
    virtual size_t size() const = 0;

    thrust::device_ptr<T> begin() {
        return thrust::device_pointer_cast<T>(data());
    }
    thrust::device_ptr<const T> begin() const {
        return thrust::device_pointer_cast<const T>(data());
    }

    thrust::device_ptr<T> end() {
        return begin() + size();
    }
    thrust::device_ptr<const T> end() const {
        return begin() + size();
    }

    thrust::device_reference<T> operator[](size_t index) {
        return thrust::device_reference<T>{begin() + index};
    }
    thrust::device_reference<const T> operator[](size_t index) const {
        return thrust::device_reference<const T>{begin() + index};
    }
};

template <typename T>
class ThrustDeviceVector : public DeviceVector<T> {
  private:
    thrust::device_vector<T> _data;

  public:
    ThrustDeviceVector(size_t size)
        : _data(size) {}

    ThrustDeviceVector(size_t size, T value)
        : _data(size, value) {}

    T* data() override {
        return thrust::raw_pointer_cast(_data.data());
    }
    T const* data() const override {
        return thrust::raw_pointer_cast(_data.data());
    }

    size_t size() const override {
        return _data.size();
    }
};

template <typename T>
class WrapperDeviceVector : public DeviceVector<T> {
  private:
    T * const _data;
    const size_t _size;

  public:
    WrapperDeviceVector(T * const data, size_t size)
        : _data(data),
          _size(size) {}

    T* data() override {
        return _data;
    }

    T const * data() const override {
        return _data;
    }

    size_t size() const override {
        return _size;
    }
};

typedef std::shared_ptr<DeviceVector<float>> device_vector_ptr;

device_vector_ptr makeThrustVector(size_t size) {
    device_vector_ptr vec = std::make_shared<ThrustDeviceVector<float>>(size);
    return vec;
}

device_vector_ptr makeThrustVector(size_t size, float value) {
    device_vector_ptr vec = std::make_shared<ThrustDeviceVector<float>>(size, value);
    return vec;
}

device_vector_ptr makeWrapperVector(float * const data, size_t size) {
	device_vector_ptr vec = std::make_shared<WrapperDeviceVector<float>>(data, size);
	return vec;
}

float* getRawPtr(device_vector_ptr& ptr) {
    return ptr->data();
}

thrust::device_ptr<float> tbegin(device_vector_ptr& vec) {
    return thrust::device_pointer_cast(vec->begin());
}

thrust::device_ptr<float> tend(device_vector_ptr& vec) {
    return thrust::device_pointer_cast(vec->end());
}

void linCombImpl(device_vector_ptr& z, float a, const device_vector_ptr& x, float b, const device_vector_ptr& y) {
    using namespace thrust::placeholders;

#if 1 //Efficient
    if (a == 0.0f) {
        if (b == 0.0f) { // z = 0
            thrust::fill(z->begin(), z->end(), 0.0f);
        } else if (b == 1.0f) { // z = y
            thrust::copy(y->begin(), y->end(), z->begin());
        } else if (b == -1.0f) { // y = -y
            thrust::transform(y->begin(), y->end(), z->begin(), -_1);
        } else { // y = b*y
            thrust::transform(y->begin(), y->end(), z->begin(), b * _1);
        }
    } else if (a == 1.0f) {
        if (b == 0.0f) { // z = x
            thrust::copy(x->begin(), x->end(), z->begin());
        } else if (b == 1.0f) { // z = x+y
            thrust::transform(x->begin(), x->end(), y->begin(), z->begin(), _1 + _2);
        } else if (b == -1.0f) { // z = x-y
            thrust::transform(x->begin(), x->end(), y->begin(), z->begin(), _1 - _2);
        } else { // z = x + b*y
            thrust::transform(x->begin(), x->end(), y->begin(), z->begin(), _1 + b * _2);
        }
    } else if (a == -1.0f) {
        if (b == 0.0f) { // z = -x
            thrust::transform(x->begin(), x->end(), z->begin(), -_1);
        } else if (b == 1.0f) { // z = -x+y
            thrust::transform(x->begin(), x->end(), y->begin(), z->begin(), -_1 + _2);
        } else if (b == -1.0f) { // z = -x-y
            thrust::transform(x->begin(), x->end(), y->begin(), z->begin(), -_1 - _2);
        } else { // z = -x + b*y
            thrust::transform(x->begin(), x->end(), y->begin(), z->begin(), -_1 + b * _2);
        }
    } else {
        if (b == 0.0f) { // z = a*x
            thrust::transform(x->begin(), x->end(), z->begin(), a * _1);
        } else if (b == 1.0f) { // z = a*x+y
            thrust::transform(x->begin(), x->end(), y->begin(), z->begin(), a * _1 + _2);
        } else if (b == -1.0f) { // z = a*x-y
            thrust::transform(x->begin(), x->end(), y->begin(), z->begin(), a * _1 - _2);
        } else { // z = a*x + b*y
            thrust::transform(x->begin(), x->end(), y->begin(), z->begin(), a * _1 + b * _2);
        }
    }
#else //Basic
    thrust::transform(x->begin(), x->end(), y->begin(), z->begin(), a * _1 + b * _2);
#endif
}

void multiplyImpl(const device_vector_ptr& v1, device_vector_ptr& v2) {
    using namespace thrust::placeholders;
    thrust::transform(v1->begin(), v1->end(), v2->begin(), v2->begin(), _1 * _2);
}

float innerImpl(const device_vector_ptr& v1, const device_vector_ptr& v2) {
    return thrust::inner_product(v1->begin(), v1->end(), v2->begin(), 0.0f);
}

//Reductions
float sumImpl(const device_vector_ptr& v) {
    return thrust::reduce(v->begin(), v->end());
}

struct Square {
    __host__ __device__ float operator()(const float& x) const { return x * x; }
};
float normImpl(const device_vector_ptr& v1) {
    return sqrtf(thrust::transform_reduce(v1->begin(), v1->end(), Square{}, 0.0f, thrust::plus<float>{}));
}

//Copies
void copyHostToDevice(double* source, device_vector_ptr& target) {
    thrust::copy_n(source, target->size(), target->begin());
}

void copyDeviceToHost(const device_vector_ptr& source, double* target) {
    thrust::copy(source->begin(), source->end(), target);
}

void printData(const device_vector_ptr& v1, std::ostream_iterator<float>& out, int numel) {
    thrust::copy(v1->begin(), v1->begin() + numel, out);
}

float getItemImpl(const device_vector_ptr& v1, int index) {
    return v1->operator[](index);
}

void setItemImpl(device_vector_ptr& v1, int index, float value) {
    v1->operator[](index) = value;
}

template <typename I1, typename I2>
void stridedGetImpl(I1 fromBegin, I1 fromEnd, I2 toBegin, int step) {
    if (step == 1) {
        thrust::copy(fromBegin, fromEnd, toBegin);
    } else {
        auto iter = make_strided_range(fromBegin, fromEnd, step);
        thrust::copy(iter.begin(), iter.end(), toBegin);
    }
}

void getSliceImpl(const device_vector_ptr& v1, int start, int stop, int step, double* target) {
    if (step > 0) {
        stridedGetImpl(v1->begin() + start, v1->begin() + stop, target, step);
    } else {
        auto reversedBegin = thrust::make_reverse_iterator(v1->begin() + start);
        auto reversedEnd = thrust::make_reverse_iterator(v1->begin() + stop);

        stridedGetImpl(reversedBegin, reversedEnd, target, -step);
    }
}

template <typename I1, typename I2>
void stridedSetImpl(I1 fromBegin, I1 fromEnd, I2 toBegin, I2 toEnd, int step) {
    if (step == 1) {
        thrust::copy(fromBegin, fromEnd, toBegin);
    } else {
        auto iter = make_strided_range(toBegin, toEnd, step);
        thrust::copy(fromBegin, fromEnd, iter.begin());
    }
}

void setSliceImpl(const device_vector_ptr& v1, int start, int stop, int step, double* source, int num) {
    if (step > 0) {
        stridedSetImpl(source, source + num, v1->begin() + start, v1->begin() + stop, step);
    } else {
        auto reversedBegin = thrust::make_reverse_iterator(v1->begin() + start);
        auto reversedEnd = thrust::make_reverse_iterator(v1->begin() + stop);

        stridedSetImpl(source, source + num, reversedBegin, reversedEnd, -step);
    }
}

__global__ void convKernel(const float* source,
                           const float* kernel,
                           float* target,
                           int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= len)
        return;

    float value = 0.0f;

    for (int i = 0; i < len; i++) {
        value += source[i] * kernel[(len + len / 2 + idx - i) % len]; //Positive modulo
    }

    target[idx] = value;
}

void convImpl(const device_vector_ptr& source, const device_vector_ptr& kernel, device_vector_ptr& target) {
    int len = source->size();
    unsigned dimBlock(256);
    unsigned dimGrid(1 + (len / dimBlock));

    convKernel<<<dimGrid, dimBlock>>>(source->data(),
                                      kernel->data(),
                                      target->data(),
                                      len);
}

// Functions
struct AbsoluteValueFunctor {
    __host__ __device__ float operator()(const float& f) { return fabs(f); }
};
void absImpl(const device_vector_ptr& source, device_vector_ptr& target) {
    thrust::transform(source->begin(), source->end(), target->begin(), AbsoluteValueFunctor{});
}

__global__ void forwardDifferenceKernel(const int len, const float* source, float* target) {
    for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < len - 1; idx += blockDim.x * gridDim.x) {
        target[idx] = source[idx + 1] - source[idx];
    }
}
void forwardDifferenceImpl(const device_vector_ptr& source, device_vector_ptr& target) {
    int len = source->size();
    unsigned dimBlock(256);
    unsigned dimGrid(std::min(128u, 1 + (len / dimBlock)));

    forwardDifferenceKernel<<<dimBlock, dimGrid>>>(len,
                                                   source->data(),
                                                   target->data());
}

__global__ void forwardDifferenceAdjointKernel(const int len, const float* source, float* target) {
    for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < len - 1; idx += blockDim.x * gridDim.x) {
        target[idx] = -source[idx] + source[idx - 1];
    }
}
void forwardDifferenceAdjointImpl(const device_vector_ptr& source, device_vector_ptr& target) {
    int len = source->size();
    unsigned dimBlock(256);
    unsigned dimGrid(std::min(128u, 1 + (len / dimBlock)));

    forwardDifferenceAdjointKernel<<<dimBlock, dimGrid>>>(len,
                                                          source->data(),
                                                          target->data());
}

void maxVectorVectorImpl(const device_vector_ptr& v1, const device_vector_ptr& v2, device_vector_ptr& target) {
    thrust::transform(v1->begin(), v1->end(), v2->begin(), target->begin(), thrust::maximum<float>{});
}

void maxVectorScalarImpl(const device_vector_ptr& source, float scalar, device_vector_ptr& target) {
    auto scalarIter = thrust::make_constant_iterator(scalar);
    thrust::transform(source->begin(), source->end(), scalarIter, target->begin(), thrust::maximum<float>{});
}

struct DivideFunctor {
    __host__ __device__ float operator()(const float& dividend, const float& divisor) { return divisor != 0.0f ? dividend / divisor : 0.0f; }
};
void divideVectorVectorImpl(const device_vector_ptr& dividend, const device_vector_ptr& divisor, device_vector_ptr& quotient) {
    thrust::transform(dividend->begin(), dividend->end(), divisor->begin(), quotient->begin(), DivideFunctor{});
}

void addScalarImpl(const device_vector_ptr& source, float scalar, device_vector_ptr& target) {
    auto scalarIter = thrust::make_constant_iterator(scalar);
    thrust::transform(source->begin(), source->end(), scalarIter, target->begin(), thrust::plus<float>{});
}

struct SignFunctor {
    __host__ __device__ float operator()(const float& f) { return (0.0f < f) - (f < 0.0f); }
};
void signImpl(const device_vector_ptr& source, device_vector_ptr& target) {
    thrust::transform(source->begin(), source->end(), target->begin(), SignFunctor{});
}
struct SqrtFunctor {
    __host__ __device__ float operator()(const float& f) { return f > 0.0f ? sqrtf(f) : 0.0f; }
};
void sqrtImpl(const device_vector_ptr& source, device_vector_ptr& target) {
    thrust::transform(source->begin(), source->end(), target->begin(), SqrtFunctor{});
}

__global__ void forwardDifference2DKernel(const int cols, const int rows, const float* data, float* dx, float* dy) {
    for (auto idy = blockIdx.y * blockDim.y + threadIdx.y + 1; idy < cols - 1; idy += blockDim.y * gridDim.y) {
        for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < rows - 1; idx += blockDim.x * gridDim.x) {
            const auto index = idx + rows * idy;

            dx[index] = data[index + 1] - data[index];
            dy[index] = data[index + rows] - data[index];
        }
    }
}

void forwardDifference2DImpl(const device_vector_ptr& source, device_vector_ptr& dx, device_vector_ptr& dy, const int cols, const int rows) {
    dim3 dimBlock(32, 32);
    dim3 dimGrid(32, 32);

    forwardDifference2DKernel<<<dimGrid, dimBlock>>>(cols, rows,
                                                     source->data(),
                                                     dx->data(),
                                                     dy->data());
}

__global__ void forwardDifference2DAdjointKernel(const int cols, const int rows, const float* dx, const float* dy, float* target) {
    for (auto idy = blockIdx.y * blockDim.y + threadIdx.y + 1; idy < cols - 1; idy += blockDim.y * gridDim.y) {
        for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < rows - 1; idx += blockDim.x * gridDim.x) {
            const auto index = idx + rows * idy;

            target[index] = -dx[index] + dx[index - 1] - dy[index] + dy[index - rows];
        }
    }
}

void forwardDifference2DAdjointImpl(const device_vector_ptr& dx, const device_vector_ptr& dy, device_vector_ptr& target, const int cols, const int rows) {
    dim3 dimBlock(32, 32);
    dim3 dimGrid(32, 32);

    forwardDifference2DAdjointKernel<<<dimGrid, dimBlock>>>(cols, rows,
                                                            dx->data(),
                                                            dy->data(),
                                                            target->data());
}
