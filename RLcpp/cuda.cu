#include "hip/hip_runtime.h"
#pragma once
#include <algorithm>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>

// thrust
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/inner_product.h>
#include <thrust/adjacent_difference.h>

// RL
#include <RLcpp/thrustUtils.h>

template <typename T>
struct uninitialized_allocator
    : thrust::device_malloc_allocator<T> {
    // note that construct is annotated as
    // a __host__ __device__ function
    __host__ __device__ void construct(T* p) {
        // no-op
    }
};

typedef thrust::device_vector<float> device_vector;
typedef std::shared_ptr<device_vector> device_vector_ptr;

device_vector_ptr makeThrustVector(size_t size) {
    return std::make_shared<device_vector>(size);
}

device_vector_ptr makeThrustVector(size_t size, float value) {
    return std::make_shared<device_vector>(size, value);
}

void linCombImpl(float a, const device_vector_ptr& x, float b, device_vector_ptr& y) {
    using namespace thrust::placeholders;

#if 1 //Efficient
    if (a == 0.0f) {
        if (b == 0.0f) { // y = 0
            thrust::fill(y->begin(), y->end(), 0.0f);
        } else if (b == 1.0f) {  // y = y, no-op
        } else if (b == -1.0f) { // y = -y
            thrust::transform(y->begin(), y->end(), y->begin(), -_1);
        } else { // y = b*y
            thrust::transform(y->begin(), y->end(), y->begin(), b * _1);
        }
    } else if (a == 1.0f) {
        if (b == 0.0f) { // y = x
            thrust::copy(x->begin(), x->end(), y->begin());
        } else if (b == 1.0f) { // y = x+y
            thrust::transform(x->begin(), x->end(), y->begin(), y->begin(), _1 + _2);
        } else if (b == -1.0f) { // y = x-y
            thrust::transform(x->begin(), x->end(), y->begin(), y->begin(), _1 - _2);
        } else { // y = x + b*y
            thrust::transform(x->begin(), x->end(), y->begin(), y->begin(), _1 + b * _2);
        }
    } else if (a == -1.0f) {
        if (b == 0.0f) { // y = -x
            thrust::transform(x->begin(), x->end(), y->begin(), -_1);
        } else if (b == 1.0f) { // y = -x+y
            thrust::transform(x->begin(), x->end(), y->begin(), y->begin(), -_1 + _2);
        } else if (b == -1.0f) { // y = -x-y
            thrust::transform(x->begin(), x->end(), y->begin(), y->begin(), -_1 - _2);
        } else { // y = -x + b*y
            thrust::transform(x->begin(), x->end(), y->begin(), y->begin(), -_1 + b * _2);
        }
    } else {
        if (b == 0.0f) { // y = a*x
            thrust::transform(x->begin(), x->end(), y->begin(), a * _1);
        } else if (b == 1.0f) { // y = a*x+y
            thrust::transform(x->begin(), x->end(), y->begin(), y->begin(), a * _1 + _2);
        } else if (b == -1.0f) { // y = a*x-y
            thrust::transform(x->begin(), x->end(), y->begin(), y->begin(), a * _1 - _2);
        } else { // y = a*x + b*y
            thrust::transform(x->begin(), x->end(), y->begin(), y->begin(), a * _1 + b * _2);
        }
    }
#else //Basic
    thrust::transform(x->begin(), x->end(), y->begin(), y->begin(), a * _1 + b * _2);
#endif
}

void multiplyImpl(const device_vector_ptr& v1, device_vector_ptr& v2) {
    using namespace thrust::placeholders;
    thrust::transform(v1->begin(), v1->end(), v2->begin(), v2->begin(), _1 * _2);
}

float innerImpl(const device_vector_ptr& v1, const device_vector_ptr& v2) {
    return thrust::inner_product(v1->begin(), v1->end(), v2->begin(), 0.0f);
}

//Reductions
float sumImpl(const device_vector_ptr& v) {
    return thrust::reduce(v->begin(), v->end());
}

struct Square {
    __host__ __device__ float operator()(const float& x) const { return x * x; }
};
float normSqImpl(const device_vector_ptr& v1) {
    return thrust::transform_reduce(v1->begin(), v1->end(), Square{}, 0.0f, thrust::plus<float>{});
}

//Copies
void copyHostToDevice(double* source, device_vector_ptr& target) {
    thrust::copy_n(source, target->size(), target->begin());
}

void copyDeviceToHost(const device_vector_ptr& source, double* target) {
    thrust::copy(source->begin(), source->end(), target);
}

void printData(const device_vector_ptr& v1, std::ostream_iterator<float>& out) {
    thrust::copy(v1->begin(), v1->end(), out);
}

float getItemImpl(const device_vector_ptr& v1, int index) {
    return v1->operator[](index);
}

void setItemImpl(device_vector_ptr& v1, int index, float value) {
    v1->operator[](index) = value;
}

void getSliceImpl(const device_vector_ptr& v1, int start, int stop, int step, double* target) {
    if (step != 1) {
		auto iter = make_strided_range(v1->begin() + start, v1->begin() + stop, step);
        thrust::copy(iter.begin(), iter.end(), target);
    } else {
        thrust::copy(v1->begin() + start, v1->begin() + stop, target);
    }
}

void setSliceImpl(const device_vector_ptr& v1, int start, int stop, int step, double* source, int num) {
    if (step != 1) {
        auto iter = make_strided_range(v1->begin() + start, v1->begin() + stop, step);
        thrust::copy(source, source + num, iter.begin());
    } else {
        thrust::copy(source, source + num, v1->begin() + start);
    }
}

__global__ void convKernel(const float* source,
                           const float* kernel,
                           float* target,
                           int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= len)
        return;

    float value = 0.0f;

    for (int i = 0; i < len; i++) {
        value += source[i] * kernel[(len + len / 2 + idx - i) % len]; //Positive modulo
    }

    target[idx] = value;
}

void convImpl(const device_vector_ptr& source, const device_vector_ptr& kernel, device_vector_ptr& target) {
    int len = source->size();
    unsigned dimBlock(256);
    unsigned dimGrid(1 + (len / dimBlock));

    convKernel<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(source->data()),
                                      thrust::raw_pointer_cast(kernel->data()),
                                      thrust::raw_pointer_cast(target->data()),
                                      len);
}

// Functions
struct AbsoluteValue {
    __host__ __device__ float operator()(const float& f) { return fabs(f); }
};
void absImpl(const device_vector_ptr& source, device_vector_ptr& target) {
    thrust::transform(source->begin(), source->end(), target->begin(), AbsoluteValue{});
}

__global__ void forwardDifferenceKernel(const int len, const float* source, float* target) {
    for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < len - 1; idx += blockDim.x * gridDim.x) {
        target[idx] = source[idx + 1] - source[idx];
    }
}
void forwardDifferenceImpl(const device_vector_ptr& source, device_vector_ptr& target) {
    int len = source->size();
    unsigned dimBlock(256);
    unsigned dimGrid(std::min(128u, 1 + (len / dimBlock)));

    forwardDifferenceKernel<<<dimBlock, dimGrid>>>(len,
                                                   thrust::raw_pointer_cast(source->data()),
                                                   thrust::raw_pointer_cast(target->data()));
}

__global__ void forwardDifferenceAdjointKernel(const int len, const float* source, float* target) {
    for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < len - 1; idx += blockDim.x * gridDim.x) {
        target[idx] = -source[idx] + source[idx - 1];
    }
}
void forwardDifferenceAdjointImpl(const device_vector_ptr& source, device_vector_ptr& target) {
    int len = source->size();
    unsigned dimBlock(256);
    unsigned dimGrid(std::min(128u, 1 + (len / dimBlock)));

    forwardDifferenceAdjointKernel<<<dimBlock, dimGrid>>>(len,
                                                          thrust::raw_pointer_cast(source->data()),
                                                          thrust::raw_pointer_cast(target->data()));
}

void maxVectorVectorImpl(const device_vector_ptr& source, device_vector_ptr& target) {
    thrust::transform(source->begin(), source->end(), target->begin(), target->begin(), thrust::maximum<float>());
}

void maxVectorScalarImpl(const device_vector_ptr& source, float scalar, device_vector_ptr& target) {
    auto scalarIter = thrust::make_constant_iterator(scalar);
    thrust::transform(source->begin(), source->end(), scalarIter, target->begin(), thrust::maximum<float>());
}

void addScalarImpl(const device_vector_ptr& source, float scalar, device_vector_ptr& target) {
    using namespace thrust::placeholders;
    thrust::transform(source->begin(), source->end(), target->begin(), _1 + scalar);
}

struct SignFunctor {
    __host__ __device__ float operator()(const float& f) { return (0.0f < f) - (f < 0.0f); }
};
void signImpl(const device_vector_ptr& source, device_vector_ptr& target) {
    thrust::transform(source->begin(), source->end(), target->begin(), SignFunctor());
}

__global__ void forwardDifference2DKernel(const int cols, const int rows, const float* data, float* dx, float* dy) {
    for (auto idy = blockIdx.y * blockDim.y + threadIdx.y + 1; idy < rows - 1; idy += blockDim.y * gridDim.y) {
        for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < cols - 1; idx += blockDim.x * gridDim.x) {
            const auto index = idx + cols * idy;

            dx[index] = data[index + 1] - data[index];
            dy[index] = data[index + cols] - data[index];
        }
    }
}
void forwardDifference2DImpl(const device_vector_ptr& source, device_vector_ptr& dx, device_vector_ptr& dy, const int cols, const int rows) {
    dim3 dimBlock(32, 32);
    dim3 dimGrid(32, 32);

    forwardDifference2DKernel<<<dimGrid, dimBlock>>>(cols, rows,
                                                     thrust::raw_pointer_cast(source->data()),
                                                     thrust::raw_pointer_cast(dx->data()),
                                                     thrust::raw_pointer_cast(dy->data()));
}

__global__ void forwardDifference2DAdjointKernel(const int cols, const int rows, const float* dx, const float* dy, float* target) {
    for (auto idy = blockIdx.y * blockDim.y + threadIdx.y + 1; idy < rows - 1; idy += blockDim.y * gridDim.y) {
        for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < cols - 1; idx += blockDim.x * gridDim.x) {
            const auto index = idx + cols * idy;

            target[index] = -dx[index] + dx[index - 1] - dy[index] + dy[index - cols];
        }
    }
}

void forwardDifference2DAdjointImpl(const device_vector_ptr& dx, const device_vector_ptr& dy, device_vector_ptr& target, const int cols, const int rows) {
    dim3 dimBlock(32, 32);
    dim3 dimGrid(32, 32);

    forwardDifference2DAdjointKernel<<<dimGrid, dimBlock>>>(cols, rows,
                                                            thrust::raw_pointer_cast(dx->data()),
                                                            thrust::raw_pointer_cast(dy->data()),
                                                            thrust::raw_pointer_cast(target->data()));
}